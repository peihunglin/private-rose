#include "hip/hip_runtime.h"
/* 
CUDA and/or nvcc does not have linker for device code.
We have to put some common device functions into this file.
So the generated CUDA file can include the functions as inlined functions.

Liao 2/11/2013
*/

/*  reduction minus is handled the same way as reduction plus since we just replace the reduction variable with its local copy for each thread
    The associated statement is intact except for the variable replacement : e.g. a-=5 becomes local_a -= 5;
    in the end of each thread accumulates thread local negative values.
    At the block level, we just simply add them all to be the block level negative values
*/
/* we have to encode the type into function name since C function signature does not include parameter list! */
#define XOMP_INNER_BLOCK_REDUCTION_DEF(dtype) \
__device__ void xomp_inner_block_reduction_##dtype(dtype local_value, dtype * grid_level_results, int reduction_op) \
{ \
  /* __shared__ float* sdata[gridDim.x]; not compilable */ \
  /* block size of data, size is specified by the kernel launch parameter (3rd one) */ \
  /* shared data has to have different names for different types. Cannot reuse name across types. */ \
  extern __shared__ dtype sdata_##dtype[];  \
  sdata_##dtype[threadIdx.x] = local_value;  \
  __syncthreads(); \
  /* blockDim.x is the block size */ \
  int isEvenSize = (blockDim.x % 2 ==0); \
  /* contiguous range pattern: half folding and add */ \
  for(int offset = blockDim.x / 2; \
      offset > 0;    /* folding and add */ \
      offset >>= 1) /* offset shrinks half each time */ \
  { \
    if(threadIdx.x < offset)  \
    { \
      /* add a partial sum upstream to our own */ \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
            sdata_##dtype[threadIdx.x] += sdata_##dtype[threadIdx.x + offset]; \
            break; \
         /*  TODO add support for more operations*/ \
         default:  \
            { \
              /* TODO: add assertion or set hipError_t with an error code */ \
              /* cannot call a host function */ \
              /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
              /* assert (false); */ \
             } \
      } /* end switch */ \
    } \
    /* remember to handle the left element */ \
    if ((threadIdx.x == 0) && !isEvenSize) \
    { \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          sdata_##dtype[0]+= sdata_##dtype[2*offset];  \
          break; \
        /* TODO add more operation support */  \
        default: \
          {  \
            /* TODO: add assertion or set hipError_t with an error code */  \
            /* cannot call a host function */ \
            /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
            /* assert (false); */ \
          } \
      } /* end switch */ \
    } \
    isEvenSize = ( offset % 2 ==0); /* prepare next round*/ \
    /* MUST wait until all threads in the block have updated their partial sums */ \
    __syncthreads(); /* sync after each folding */ \
  } \
  /* thread 0 writes the final result to the partial sum of this thread block */ \
  if(threadIdx.x == 0) \
  { \
    grid_level_results[blockIdx.x] = sdata_##dtype[0]; \
  } \
}

XOMP_INNER_BLOCK_REDUCTION_DEF(int)
XOMP_INNER_BLOCK_REDUCTION_DEF(float)
XOMP_INNER_BLOCK_REDUCTION_DEF(double)

#undef XOMP_INNER_BLOCK_REDUCTION_DEF 

