#include "hip/hip_runtime.h"
/* A simple lib to facilitate CUDA coding 

Extensions to the xomp interface

Liao 4/11/2012
*/

//#include "cuda_lib.h"
#include "libxomp.h"
//----------------------------------------------------
// Device xomp_cuda_property retrieving functions


hipDeviceProp_t* xomp_cuda_prop = NULL; 
bool xomp_verbose = false;

// this can be called multiple times. But the xomp_cuda_prop variable will only be set once
hipDeviceProp_t * xomp_getCudaDeviceProp()
{
  if (xomp_cuda_prop == NULL )
  {
    xomp_cuda_prop = (hipDeviceProp_t *) malloc(sizeof(hipDeviceProp_t));
    assert (xomp_cuda_prop != NULL);
    int count;
    hipGetDeviceCount (&count);
    assert (count>=1); // must have at least one GPU here
    hipGetDeviceProperties  (xomp_cuda_prop, 0);
  }
  return xomp_cuda_prop;
}

void xomp_print_gpu_info()
{
  int max_threads_per_block = xomp_getCudaDeviceProp()->maxThreadsPerBlock;
  int max_blocks_per_grid_x = xomp_getCudaDeviceProp()->maxGridSize[0];
  int global_memory_size =    xomp_getCudaDeviceProp()->totalGlobalMem;
  int shared_memory_size =    xomp_getCudaDeviceProp()->sharedMemPerBlock;
  int registers_per_block =   xomp_getCudaDeviceProp()->regsPerBlock;

  printf ("Found a GPU with \n\tmax threads per block=%d, \n\tmax blocks for Grid X dimension=%d\n\
      \tglobal mem bytes =%d, \n\tshared mem bytes =%d, \n\tregs per block = %d\n",
      max_threads_per_block, max_blocks_per_grid_x, global_memory_size ,  shared_memory_size,
      registers_per_block);
}

size_t xomp_get_max1DBlock(size_t s)
{
  size_t block_num = s/xomp_get_maxThreadsPerBlock();
  if (s % xomp_get_maxThreadsPerBlock()!= 0)
     block_num ++;
  return block_num;     
}

// Get the max number threads for one dimension (x or y) of a 2D block
// Two factors are considered: the total number of threads within the 2D block must<= total threads per block
//  x * y <= maxThreadsPerBlock 512 or 1024
// each dimension: the number of threads must <= maximum x/y-dimension
//    x <= maxThreadsDim[0],  1024
//    y <= maxThreadsDim[1], 1024 
//  maxThreadsDim[0] happens to be equal to  maxThreadsDim[1] so we use a single function to calculate max segments for both dimensions
size_t xomp_get_max_threads_per_dimesion_2D ()
{

  int max_threads_per_block = xomp_getCudaDeviceProp()->maxThreadsPerBlock;
  // we equalize the number of threads in each dimension
  int max_threads_per_2d_dimension = (int)(sqrt((float)max_threads_per_block));  
  assert (max_threads_per_2d_dimension*max_threads_per_2d_dimension<= max_threads_per_block);

  // our assumption is that dim[0] == dim[1] so we handle x and y in one function
  assert ( xomp_getCudaDeviceProp()->maxThreadsDim[0] == xomp_getCudaDeviceProp()->maxThreadsDim[1]);   
  assert (max_threads_per_2d_dimension <= xomp_getCudaDeviceProp()->maxThreadsDim[0]);
  return max_threads_per_2d_dimension;
}

// return the max number of segments for a dimension (either x or y) of a 2D block
// we define the number of segments to be  SIZE_of_Dimension_x/max_threads_x_dimension
size_t xomp_get_maxSegmentsPerDimensionOf2DBlock(size_t dimension_size)
{
  // For simplicity, we don't yet consider the factor of warp size for now
  // TODO: block size should be divisible by the warp size??
  // e.g. max threads per block is 1024, then max number of tiles per dimension in a 2D block is 1024^0.5 = 32 threads
  size_t max_threads_per_2d_dimension = xomp_get_max_threads_per_dimesion_2D ();
  size_t block_num_x_or_y =  dimension_size/max_threads_per_2d_dimension;
  if (dimension_size % max_threads_per_2d_dimension != 0)
     block_num_x_or_y ++;

  return block_num_x_or_y;
}


// max thread per block, useful for 1-D problem
size_t xomp_get_maxThreadsPerBlock()
{
  return xomp_getCudaDeviceProp()->maxThreadsPerBlock;
}

/*-----------------------------------------------------
   Device memory allocation functions 
*/
void* xomp_deviceMalloc(size_t size)
{
  void * devPtr;
  hipError_t rt =  hipMalloc(&devPtr, size);
  if ( (size !=0) && (rt == hipSuccess) )
  {
    return devPtr;
  }
  else
  {
    fprintf(stderr, "Error: hipMalloc() failed to allocate the requested %d bytes!\n",size );
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

// A host version
void* xomp_hostMalloc(size_t size)
{
  assert (size>0);
  void* hostPtr;
  hostPtr = (char*) malloc (size);
  if (hostPtr == NULL)
  {
    fprintf(stderr, "Error: malloc() failed to allocate the requested %d bytes!\n",size );
    assert (hostPtr != NULL);
  }
  return hostPtr; 
}

// memory copy from src to dest, return the pointer to dest. NULL pointer if anything is wrong 
void * xomp_memcpyHostToDevice (void *dest, const void * src, size_t n)
{
  assert (dest != NULL);
  assert (src != NULL);
  if (xomp_verbose)
    printf("xomp_memcpyHostToDevice(): dest=%p src =%p size=%d\n",dest, src, n);
  hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyHostToDevice);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy memory from Host %p to Device %p, for %d bytes!\n",src, dest, n);
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

void * xomp_memcpyDeviceToHost (void *dest, const void * src, size_t n)
{
  assert (dest != NULL);
  assert (src != NULL);
  if (xomp_verbose)
    printf("xomp_memcpyDeviceToHost(): dest=%p src =%p size=%d\n",dest, src, n);
  hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyDeviceToHost);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy memory from Device %p to Host %p, for %d bytes!\n",src, dest, n);
    fprintf(stderr, "Error message is =%s\n",hipGetErrorString(rt));
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

// copy a dynamically allocated host source array to a linear dest address on a GPU device. 
// The dimension information of the source array is given by: int dimensions[dimension_size], with known element size. 
// bytes_copied reports the total bytes copied by this function.  
// Liao 4/25/2012
void * xomp_memcpyDynamicHostToDevice (void *dest, const void * src, int * dimensions, size_t dimension_size, size_t element_size, size_t *bytes_copied)
{
  assert (dest != NULL);
  assert (src != NULL);
  if  (dimension_size == 1)  // down to the final, inner-most dimension 
  {
    // this is a firm, bottom count of bytes copied
    *bytes_copied = element_size* dimensions[0]; // number of elements * element_size
    xomp_memcpyHostToDevice (dest, src, *bytes_copied);
  }
  else 
  { 
    assert (dimension_size>=2);
    // 2-D or more: arrays of sub-arrays, copy each sub-array separately
    void ** array2 = (void**) src ; // re-interpret src to be array of arrays
    size_t sub_array_count = dimensions[0];  // top dimension, how many sub-arrays

    // prepare dimension information for sub-arrays
    dimensions ++;// go to the next dimension
    dimension_size --; // sub-array has one less dimension
    char* new_dest = (char*) dest; // byte addressable for dest
    size_t total_subarray_bytes_copied =0; 
    int i; 
    for (i = 0; i< sub_array_count; i++) // for each sub-arrays
    {
      size_t subarray_bytes_copied = 0; 
      void* sub_array_src = (void*) (array2[i]);  // get start address for each sub-array
      assert (sub_array_src != NULL);
      // recursively call to copy each sub-array
      xomp_memcpyDynamicHostToDevice (new_dest, (void*)sub_array_src, dimensions, dimension_size, 
          element_size, &subarray_bytes_copied);

      total_subarray_bytes_copied += subarray_bytes_copied; // recursively accumulate the bytes copied, instead of calculating directly.
      new_dest += subarray_bytes_copied ; // update the dest offset here
    }
    *bytes_copied = total_subarray_bytes_copied; 
  }
  return dest; 
}


// copy linear src memory to dynamically allocated destination, with dimension information given by
// int dimensions[dimension_size]
// the source memory has total n continuous memory, with known size for each element
// the total bytes copied by this function is reported by bytes_copied
// test code: mallocArray-xomp.cu
void * xomp_memcpyDynamicDeviceToHost (void *dest, int * dimensions, size_t dimension_size, const void * src, size_t element_size, size_t *bytes_copied)
{
  // int dimensions[1] = {10}; dimension_size =1; 
  // a[10]: 
  if  (dimension_size == 1)  // down to the final dimension 
  {
    // this is a firm, bottom count of bytes copied
    *bytes_copied = element_size* dimensions[0]; // number of elements * element_size
    xomp_memcpyDeviceToHost (dest, src, *bytes_copied);
  }
  else 
  { 
    int i; 
    assert (dimension_size>=2);
    // 2-D or more: arrays of sub-arrays, copy each sub-array separately
    void ** array2 = (void**) dest; // re-interpret dest to be array of arrays
    size_t sub_array_count = dimensions[0];  // current dimension, how many sub-arrays

    // prepare dimension information for sub-arrays
    dimensions ++;// go to the next dimension
    dimension_size --;
    char* new_src = (char*) src; // byte addressable for src
    size_t total_subarray_bytes_copied =0; 
    for (i = 0; i< sub_array_count; i++) // for each sub-arrays
    {
      size_t subarray_bytes_copied =0; 
      void* sub_array_dest = (void*) (array2[i]);  // get start address for each sub-array

      xomp_memcpyDynamicDeviceToHost ((void*)sub_array_dest,  dimensions, dimension_size, 
          new_src, element_size, &subarray_bytes_copied);

      total_subarray_bytes_copied += subarray_bytes_copied; // recursively accumulate the bytes copied, instead of calculating directly.
      new_src += subarray_bytes_copied ; // update the source offset here
    }
    *bytes_copied = total_subarray_bytes_copied; 
  }
  return dest; 
}

void * xomp_memcpyDeviceToDevice (void *dest, const void * src, size_t n)
{
    hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyDeviceToDevice);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy from Device %p to Device %p for requested %d bytes!\n", src, dest, n );
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

void * xomp_memcpyHostToHost (void *dest, const void * src, size_t n) // same as memcpy??
{
    hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyHostToHost);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy from Host %p to Host %p for requested %d bytes!\n", src, dest, n );
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }

}

//------------------------------------------------------
// free the device memory pointed by a pointer, return false in case of failure, otherwise return true
bool xomp_freeDevice(void* devPtr)
{
  hipError_t rt = hipFree (devPtr);
  if (rt == hipSuccess)
    return true;
  else
    return false;
}

// free the host memory pointed by a pointer, return false in case of failure, otherwise return true
bool xomp_freeHost(void* hostPtr)
{
  hipError_t rt = hipHostFree(hostPtr);
  if (rt == hipSuccess)
    return true;
  else
    return false;
}

//------------------------------------------------------
// data set size checking functions
#if 0
  // make sure the length of the array can be mapped to the cuda threads
  assert (SIZE <= max_blocks_per_grid_x* max_threads_per_block);
  // make sure the data will fit into the device memory (shared memory)
  printf("matrix-vector multiplication with size=%d\n", SIZE);
  // one matrix and two vectors
  int mem_required = SIZE*SIZE*sizeof(float) + SIZE* sizeof(float) *2;
  if (global_memory_size > 0) //sometimes the number is too large and it overflows to be a negative integer
    assert (mem_required  <= global_memory_size);
#endif

//------------------------------------------------------
double xomp_time_stamp()
{
  struct timeval t;
  double time;
  gettimeofday(&t, NULL);
  time = t.tv_sec + 1.0e-6*t.tv_usec;
  return time;
}

//------------------------------------------------------
// Host side helper functions
//--- a helper function to allocate 2-D arrays
/* Allocate a multi-dimensional array
 *
 * Input parameters:
 *  int *dimensions:  an integer array storing the size of each dimension
 *  size_t dimension_num: the number of dimensions
 *  size_t esize: the size of an array element
 *
 * return:
 *  the pointer to the allocated array
 * */
void * xomp_mallocArray(int * dimensions, size_t dimension_num, size_t esize)
{
  int i;
  void * array = NULL;

//  if (xomp_verbose)
//    printf("xomp_xomp_mallocArray(): dimensions=%p dimension =%d element size=%d\n",dimensions, dimension_num, esize);
  //  Handle 1-D array: do element-wise malloc
  if ( dimension_num == 1)
  {
    array = (void *) malloc(dimensions[dimension_num - 1]*esize);
    if(array == NULL)
    {
      fprintf(stderr, "out of memory\n");
      abort();
    }
  }
  else // two and more dimensions to be allocated: reduce it to be a 2-step allocation
  {
    // 1st step: allocate the first dimension 
    //           by treating it as allocating a 1-D array of arrays (pointer)
    void ** array2 = NULL;
    array2 = (void **) xomp_mallocArray(dimensions, 1 ,sizeof (void *));
    size_t prev_dim_size = dimensions[0];// number of elements of the first dimension

    // 2nd step: allocate the remaining N -1 dimension arrays, each is an element of the first array 
    // peel off the 1st(previous) dimension, focus on the rest dimensions
     dimensions ++;
    // each of element is an array has a smaller dimension number
     dimension_num --;
    for(i = 0; i < prev_dim_size ; i++)
    {
      array2[i] = xomp_mallocArray (dimensions, dimension_num, esize);
    }
    // return the pointer to the first dimension
    array = (void *) array2;
  }
  return array;
}


/* Free a pointer to a multi-dimensional array
 * int * dimensions: store the sizes of each dimension
 * size_t dimension_num: the number of dimensions
 *
 * */
void xomp_freeArrayPointer (void* array, int * dimensions, size_t dimension_num)
{

  int i;
  // 1-D case, call free() directly
  if (dimension_num == 1)
  {
    free (array);
  }
  else
  { // 2-D or more, iterate through higher dimension and try to free inner arrays
    int prev_dim_size = dimensions [0];
    // step into one dimension
    dimensions ++;
    dimension_num --;
    for (i =0; i< prev_dim_size ; i++)
    {
      xomp_freeArrayPointer (((void **)array)[i], dimensions, dimension_num);
    }
  }
}

#if 0
/*  reduction minus is handled the same way as reduction plus since we just replace the reduction variable with its local copy for each thread
    The associated statement is intact except for the variable replacement : e.g. a-=5 becomes local_a -= 5;
    in the end of each thread accumulates thread local negative values.
    At the block level, we just simply add them all to be the block level negative values
*/
/* we have to encode the type into function name since C function signature does not include parameter list! */
#define XOMP_INNER_BLOCK_REDUCTION_DEF(dtype) \
__device__ void xomp_inner_block_reduction_##dtype(dtype local_value, dtype * grid_level_results, int reduction_op) \
{ \
  /* __shared__ float* sdata[gridDim.x]; not compilable */ \
  /* block size of data, size is specified by the kernel launch parameter (3rd one) */ \
  /* shared data has to have different names for different types. Cannot reuse name across types. */ \
  extern __shared__ dtype sdata_##dtype[];  \
  sdata_##dtype[threadIdx.x] = local_value;  \
  __syncthreads(); \
  /* blockDim.x is the block size */ \
  int isEvenSize = (blockDim.x % 2 ==0); \
  /* contiguous range pattern: half folding and add */ \
  for(int offset = blockDim.x / 2; \
      offset > 0;    /* folding and add */ \
      offset >>= 1) /* offset shrinks half each time */ \
  { \
    if(threadIdx.x < offset)  \
    { \
      /* add a partial sum upstream to our own */ \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
            sdata_##dtype[threadIdx.x] += sdata_##dtype[threadIdx.x + offset]; \
            break; \
         /*  TODO add support for more operations*/ \
         default:  \
            { \
              /* TODO: add assertion or set hipError_t with an error code */ \
              /* cannot call a host function */ \
              /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
              /* assert (false); */ \
             } \
      } /* end switch */ \
    } \
    /* remember to handle the left element */ \
    if ((threadIdx.x == 0) && !isEvenSize) \
    { \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          sdata_##dtype[0]+= sdata_##dtype[2*offset];  \
          break; \
        /* TODO add more operation support */  \
        default: \
          {  \
            /* TODO: add assertion or set hipError_t with an error code */  \
            /* cannot call a host function */ \
            /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
            /* assert (false); */ \
          } \
      } /* end switch */ \
    } \
    isEvenSize = ( offset % 2 ==0); /* prepare next round*/ \
    /* MUST wait until all threads in the block have updated their partial sums */ \
    __syncthreads(); /* sync after each folding */ \
  } \
  /* thread 0 writes the final result to the partial sum of this thread block */ \
  if(threadIdx.x == 0) \
  { \
    grid_level_results[blockIdx.x] = sdata_##dtype[0]; \
  } \
}

XOMP_INNER_BLOCK_REDUCTION_DEF(int)
XOMP_INNER_BLOCK_REDUCTION_DEF(float)
XOMP_INNER_BLOCK_REDUCTION_DEF(double)

#undef XOMP_INNER_BLOCK_REDUCTION_DEF 

#endif

// TODO: handle more different reduction operations
// TODO : add assertion support
#define XOMP_BEYOND_BLOCK_REDUCTION_DEF(dtype) \
  dtype xomp_beyond_block_reduction_##dtype(dtype * per_block_results, int numBlocks, int reduction_op) \
{ \
  dtype result ; \
  dtype* per_block_results_cpu = (dtype *)xomp_hostMalloc (numBlocks*sizeof(dtype)); \
  xomp_memcpyDeviceToHost (per_block_results_cpu, per_block_results, sizeof(dtype)* numBlocks); \
  int r_i; \
  for (r_i =1; r_i < numBlocks; r_i++) \
  { \
     switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          per_block_results_cpu[0]+= per_block_results_cpu[r_i]; \
          break; \
        default: \
          { \
          } \
      } \
  } \
  result = per_block_results_cpu[0]; \
  xomp_freeHost(per_block_results_cpu); \
  return result; \
} 

//TODO define more types of CPU level reduction support
XOMP_BEYOND_BLOCK_REDUCTION_DEF(int)
XOMP_BEYOND_BLOCK_REDUCTION_DEF(float)
XOMP_BEYOND_BLOCK_REDUCTION_DEF(double)

#undef XOMP_BEYOND_BLOCK_REDUCTION_DEF 






