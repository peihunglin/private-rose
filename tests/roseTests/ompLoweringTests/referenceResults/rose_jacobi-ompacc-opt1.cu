#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#ifdef _OPENMP
#include <omp.h>
#endif
// Add timing support
#include <sys/time.h>
#include "libxomp.h" 
#include "xomp_cuda_lib_inlined.cu" 

double time_stamp()
{
  struct timeval t;
  double time;
  gettimeofday(&t,0);
  time = (t.tv_sec + (1.0e-6 * t.tv_usec));
  return time;
}
double time1;
double time2;
void driver();
void initialize();
void jacobi();
void error_check();
/************************************************************
* program to solve a finite difference 
* discretization of Helmholtz equation :  
* (d2/dx2)u + (d2/dy2)u - alpha u = f 
* using Jacobi iterative method. 
*
* Modified: Sanjiv Shah,       Kuck and Associates, Inc. (KAI), 1998
* Author:   Joseph Robicheaux, Kuck and Associates, Inc. (KAI), 1998
*
* This c version program is translated by 
* Chunhua Liao, University of Houston, Jan, 2005 
* 
* Directives are used in this code to achieve parallelism. 
* All do loops are parallelized with default 'static' scheduling.
* 
* Input :  n - grid dimension in x direction 
*          m - grid dimension in y direction
*          alpha - Helmholtz constant (always greater than 0.0)
*          tol   - error tolerance for iterative solver
*          relax - Successice over relaxation parameter
*          mits  - Maximum iterations for iterative solver
*
* On output 
*       : u(n,m) - Dependent variable (solutions)
*       : f(n,m) - Right hand side function 
*************************************************************/
#define MSIZE 512
int n;
int m;
int mits;
#define REAL float // flexible between float and double
float tol;
float relax = 1.0;
float alpha = 0.0543;
float u[512UL][512UL];
float f[512UL][512UL];
float uold[512UL][512UL];
float dx;
float dy;

int main()
{
//  float toler;
/*      printf("Input n,m (< %d) - grid dimension in x,y direction:\n",MSIZE); 
          scanf ("%d",&n);
          scanf ("%d",&m);
          printf("Input tol - error tolerance for iterative solver\n"); 
          scanf("%f",&toler);
          tol=(double)toler;
          printf("Input mits - Maximum iterations for solver\n"); 
          scanf("%d",&mits);
          */
  n = 512;
  m = 512;
  tol = 0.0000000001;
  mits = 5000;
#if 0 // Not yet support concurrent CPU and GPU threads  
#ifdef _OPENMP
#endif
#endif  
  driver();
  return 0;
}
/*************************************************************
* Subroutine driver () 
* This is where the arrays are allocated and initialzed. 
*
* Working varaibles/arrays 
*     dx  - grid spacing in x direction 
*     dy  - grid spacing in y direction 
*************************************************************/

void driver()
{
  initialize();
  time1 = time_stamp();
/* Solve Helmholtz equation */
  jacobi();
  time2 = time_stamp();
  printf("------------------------\n");
  printf("Execution time = %f\n",(time2 - time1));
/* error_check (n,m,alpha,dx,dy,u,f)*/
  error_check();
}
/*      subroutine initialize (n,m,alpha,dx,dy,u,f) 
******************************************************
* Initializes data 
* Assumes exact solution is u(x,y) = (1-x^2)*(1-y^2)
*
******************************************************/

void initialize()
{
  int i;
  int j;
  int xx;
  int yy;
//double PI=3.1415926;
  dx = (2.0 / (n - 1));
  dy = (2.0 / (m - 1));
/* Initialize initial condition and RHS */
//#pragma omp parallel for private(xx,yy,j,i)
  for (i = 0; i < n; i++) 
    for (j = 0; j < m; j++) {
      xx = ((int )(-1.0 + (dx * (i - 1))));
      yy = ((int )(-1.0 + (dy * (j - 1))));
      u[i][j] = 0.0;
      f[i][j] = (((((-1.0 * alpha) * (1.0 - (xx * xx))) * (1.0 - (yy * yy))) - (2.0 * (1.0 - (xx * xx)))) - (2.0 * (1.0 - (yy * yy))));
    }
}
/*      subroutine jacobi (n,m,dx,dy,alpha,omega,u,f,tol,maxit)
******************************************************************
* Subroutine HelmholtzJ
* Solves poisson equation on rectangular grid assuming : 
* (1) Uniform discretization in each direction, and 
* (2) Dirichlect boundary conditions 
* 
* Jacobi method is used in this routine 
*
* Input : n,m   Number of grid points in the X/Y directions 
*         dx,dy Grid spacing in the X/Y directions 
*         alpha Helmholtz eqn. coefficient 
*         omega Relaxation factor 
*         f(n,m) Right hand side function 
*         u(n,m) Dependent variable/Solution
*         tol    Tolerance for iterative solver 
*         maxit  Maximum number of iterations 
*
* Output : u(n,m) - Solution 
*****************************************************************/

__global__ void OUT__1__10550__(int n,int m,float omega,float ax,float ay,float b,float *_dev_per_block_error,float *_dev_u,float *_dev_f,float *_dev_uold)
{
  int _p_i;
  int _p_j;
  float _p_error;
  _p_error = 0;
  float _p_resid;
  int _dev_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (_dev_i >= 1 && _dev_i <= (n - 1) - 1) {
    for (_p_j = 1; _p_j < (m - 1); _p_j++) {
      _p_resid = (((((ax * (_dev_uold[(_dev_i - 1) * 512UL + _p_j] + _dev_uold[(_dev_i + 1) * 512UL + _p_j])) + (ay * (_dev_uold[_dev_i * 512UL + (_p_j - 1)] + _dev_uold[_dev_i * 512UL + (_p_j + 1)]))) + (b * _dev_uold[_dev_i * 512UL + _p_j])) - _dev_f[_dev_i * 512UL + _p_j]) / b);
      _dev_u[_dev_i * 512UL + _p_j] = (_dev_uold[_dev_i * 512UL + _p_j] - (omega * _p_resid));
      _p_error = (_p_error + (_p_resid * _p_resid));
    }
  }
  xomp_inner_block_reduction_float(_p_error,_dev_per_block_error,6);
}

__global__ void OUT__2__10550__(int n,int m,float *_dev_u,float *_dev_uold)
{
  int _p_i;
  int _p_j;
  int _dev_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (_dev_i >= 0 && _dev_i <= n - 1) {
    for (_p_j = 0; _p_j < m; _p_j++) 
      _dev_uold[_dev_i * 512UL + _p_j] = _dev_u[_dev_i * 512UL + _p_j];
  }
}

void jacobi()
{
  float omega;
  int i;
  int j;
  int k;
  float error;
  float resid;
  float ax;
  float ay;
  float b;
//      double  error_local;
//      float ta,tb,tc,td,te,ta1,ta2,tb1,tb2,tc1,tc2,td1,td2;
//      float te1,te2;
//      float second;
  omega = relax;
/*
   * Initialize coefficients */
/* X-direction coef */
  ax = (1.0 / (dx * dx));
/* Y-direction coef */
  ay = (1.0 / (dy * dy));
/* Central coeff */
  b = (((-2.0 / (dx * dx)) - (2.0 / (dy * dy))) - alpha);
  error = (10.0 * tol);
  k = 1;
/* Translated from #pragma omp target data ... */
{
    float *_dev_u;
    int _dev_u_size = sizeof(float ) * (n - 0) * (m - 0);
    _dev_u = ((float *)(xomp_deviceMalloc(_dev_u_size)));
    xomp_memcpyHostToDevice(((void *)_dev_u),((const void *)u),_dev_u_size);
    float *_dev_f;
    int _dev_f_size = sizeof(float ) * (n - 0) * (m - 0);
    _dev_f = ((float *)(xomp_deviceMalloc(_dev_f_size)));
    xomp_memcpyHostToDevice(((void *)_dev_f),((const void *)f),_dev_f_size);
    float *_dev_uold;
    int _dev_uold_size = sizeof(float ) * (n - 0) * (m - 0);
    _dev_uold = ((float *)(xomp_deviceMalloc(_dev_uold_size)));
    while((k <= mits) && (error > tol)){
      error = 0.0;
/* Copy new solution into old */
{
/* Launch CUDA kernel ... */
        int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
        int _num_blocks_ = xomp_get_max1DBlock(n - 1 - 0 + 1);
        OUT__2__10550__<<<_num_blocks_,_threads_per_block_>>>(n,m,_dev_u,_dev_uold);
      }
{
/* Launch CUDA kernel ... */
        int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
        int _num_blocks_ = xomp_get_max1DBlock((n - 1) - 1 - 1 + 1);
        float *_dev_per_block_error = (float *)(xomp_deviceMalloc(_num_blocks_ * sizeof(float )));
        OUT__1__10550__<<<_num_blocks_,_threads_per_block_,(_threads_per_block_ * sizeof(float ))>>>(n,m,omega,ax,ay,b,_dev_per_block_error,_dev_u,_dev_f,_dev_uold);
        error = xomp_beyond_block_reduction_float(_dev_per_block_error,_num_blocks_,6);
        xomp_freeDevice(_dev_per_block_error);
      }
//    }
/*  omp end parallel */
/* Error check */
      if ((k % 500) == 0) 
        printf("Finished %d iteration with error =%f\n",k,error);
      error = (sqrt(error) / (n * m));
      k = (k + 1);
/*  End iteration loop */
    }
    xomp_memcpyDeviceToHost(((void *)u),((const void *)_dev_u),_dev_u_size);
    xomp_freeDevice(_dev_u);
    xomp_freeDevice(_dev_f);
    xomp_freeDevice(_dev_uold);
  }
  printf("Total Number of Iterations:%d\n",k);
  printf("Residual:%E\n",error);
}
/*      subroutine error_check (n,m,alpha,dx,dy,u,f) 
      implicit none 
************************************************************
* Checks error between numerical and exact solution 
*
************************************************************/

void error_check()
{
  int i;
  int j;
  float xx;
  float yy;
  float temp;
  float error;
  dx = (2.0 / (n - 1));
  dy = (2.0 / (m - 1));
  error = 0.0;
//#pragma omp parallel for private(xx,yy,temp,j,i) reduction(+:error)
  for (i = 0; i < n; i++) 
    for (j = 0; j < m; j++) {
      xx = (-1.0 + (dx * (i - 1)));
      yy = (-1.0 + (dy * (j - 1)));
      temp = (u[i][j] - ((1.0 - (xx * xx)) * (1.0 - (yy * yy))));
      error = (error + (temp * temp));
    }
  error = (sqrt(error) / (n * m));
  printf("Solution Error :%E \n",error);
}
