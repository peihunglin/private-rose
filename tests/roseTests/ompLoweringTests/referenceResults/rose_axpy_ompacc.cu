#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
// Liao 1/15/2013
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
#include "libxomp.h" 
#include "xomp_cuda_lib_inlined.cu" 

void zero(double *A,int n)
{
  int i;
  for (i = 0; i < n; i++) {
    A[i] = 0.0;
  }
}
/* initialize a vector with random floating point numbers */

void init(double *A,int n)
{
  int i;
  for (i = 0; i < n; i++) {
    A[i] = drand48();
  }
}

__global__ void OUT__1__10001__(int n,double a,double *_dev_x,double *_dev_y)
{
  int _p_i;
  int _dev_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (_dev_i >= 0 && _dev_i <= n - 1) {
    _dev_y[_dev_i] += (a * _dev_x[_dev_i]);
  }
}

void axpy_ompacc(double *x,double *y,int n,double a)
{
  int i;
/* this one defines both the target device name and data environment to map to,
   I think here we need mechanism to tell the compiler the device type (could be multiple) so that compiler can generate the codes of different versions; 
   we also need to let the runtime know what the target device is so the runtime will chose the right function to call if the code are generated 
   #pragma omp target device (gpu0) map(x, y) 
*/
{
    double *_dev_x;
    int _dev_x_size = sizeof(double ) * (n - 0);
    _dev_x = ((double *)(xomp_deviceMalloc(_dev_x_size)));
    xomp_memcpyHostToDevice(((void *)_dev_x),((const void *)x),_dev_x_size);
    double *_dev_y;
    int _dev_y_size = sizeof(double ) * (n - 0);
    _dev_y = ((double *)(xomp_deviceMalloc(_dev_y_size)));
    xomp_memcpyHostToDevice(((void *)_dev_y),((const void *)y),_dev_y_size);
/* Launch CUDA kernel ... */
    int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
    int _num_blocks_ = xomp_get_max1DBlock(n - 1 - 0 + 1);
    OUT__1__10001__<<<_num_blocks_,_threads_per_block_>>>(n,a,_dev_x,_dev_y);
    xomp_freeDevice(_dev_x);
    xomp_memcpyDeviceToHost(((void *)y),((const void *)_dev_y),_dev_y_size);
    xomp_freeDevice(_dev_y);
  }
}

int main(int argc,char *argv[])
{
  int n;
  double *y_ompacc;
  double *x;
  double a = 123.456;
  n = 1024000;
  y_ompacc = ((double *)(malloc((n * sizeof(double )))));
  x = ((double *)(malloc((n * sizeof(double )))));
  srand48((1 << 12));
  init(x,n);
  init(y_ompacc,n);
/* openmp acc version */
  axpy_ompacc(x,y_ompacc,n,a);
  free(y_ompacc);
  free(x);
  return 0;
}
